#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>

__global__ void linear_forward_bf16(const hip_bfloat16* input, const hip_bfloat16* weight, const hip_bfloat16* bias, hip_bfloat16* output,
                                    int input_size, int output_size, int batch_size) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < batch_size && col < output_size) {
        float sum = 0.0f;
        for (int i = 0; i < input_size; ++i) {
            sum += __bfloat162float(input[row * input_size + i]) * __bfloat162float(weight[col * input_size + i]);
        }
        output[row * output_size + col] = bias ? __float2bfloat16(sum + __bfloat162float(bias[col])) : __float2bfloat16(sum);
    }
}


void linear_forward_cuda_bf16(torch::Tensor input, torch::Tensor weight, torch::Tensor bias, torch::Tensor output) {
    int batch_size = input.size(0);
    int input_size = input.size(1);
    int output_size = weight.size(0);

    const dim3 blocks(batch_size);
    const dim3 threads(output_size);

    AT_CUDA_CHECK(hipGetLastError());
    linear_forward_bf16<<<blocks, threads>>>(
        reinterpret_cast<hip_bfloat16*>(input.data_ptr()),
        reinterpret_cast<hip_bfloat16*>(weight.data_ptr()),
        bias.defined() ? reinterpret_cast<hip_bfloat16*>(bias.data_ptr()) : nullptr,
        reinterpret_cast<hip_bfloat16*>(output.data_ptr()),
        input_size, output_size, batch_size);
    AT_CUDA_CHECK(hipGetLastError());
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("linear_forward_bf16", &linear_forward_cuda_bf16, "Linear forward with bf16 (CUDA)");
}
